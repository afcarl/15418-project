#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

void copy_data(double* input, int n, int p) {
    int* device_input;

    hipMalloc((void **)&device_input, sizeof(double) * n * p);
    hipMemcpy(device_input, input, sizeof(double) * n * p, hipMemcpyHostToDevice);
    hipFree(device_input);

}
