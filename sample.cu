#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>


#include "CycleTimer.h"

#define N (513)
#define P (9)
#define SQRT_P (3)
#define NUM_BLOCKS (3)
#define THREADS_PER_BLOCK 1024
#define BLOCKSIZE (1024)
#define UPDIV(N, threadsPerBlock) (((N) + (threadsPerBlock) - 1) / (threadsPerBlock))


// From http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
__global__ void init(unsigned int seed, hiprandState_t* states) {
     hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

__global__ void randoms(hiprandState_t* states, int* numbers) {
    numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % N;
}

__global__ void make_sample(double* in_data,
                            double* out_data,
                            int* sample_idx,
                            int n,
                            int p) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) {
        return;
    }

    for (int i = 0; i < p; i++) {
        int row_idx = sample_idx[idx];
        out_data[idx * p + i] = in_data[row_idx * p + i];
    }
}

void bootstrap_sample(double* in_data, double** out_data, int n, int p) {
    int* device_idx;
    hiprandState_t* states;

    hipMalloc((void**) &device_idx, sizeof(int) * n);
    hipMalloc((void**) &states, sizeof(hiprandState_t) * n);

    init<<<N, 1>>>(time(0), states);
    randoms<<<N, 1>>>(states, device_idx);


    const int blocks = UPDIV(N, THREADS_PER_BLOCK);
    make_sample<<<blocks, THREADS_PER_BLOCK>>>(in_data, *out_data, device_idx, n, p);

    hipFree(device_idx);
}

void bootstrap_sample(int** device_nums) {
    hipMalloc(device_nums, sizeof(int) * N);
    hiprandState_t* states;
    hipMalloc((void**) &states, sizeof(hiprandState_t) * N);

    init<<<N, 1>>>(time(0), states);
    randoms<<<N, 1>>>(states, *device_nums);

    hipFree(states);
}

