#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

__global__ void split() {
}

void data_to_device(double* device_input_data, double* input_data, int size) {
    hipMalloc((void **)&device_input_data, sizeof(double) * size);
    hipMemcpy(device_input_data, input_data, sizeof(double) * size, hipMemcpyHostToDevice);
}

void data_to_device(int* device_input_data, int* input_data, int size) {
    hipMalloc((void **)&device_input_data, sizeof(int) * size);
    hipMemcpy(device_input_data, input_data, sizeof(int) * size, hipMemcpyHostToDevice);
}

void free_from_device(int* device_data) {
    hipFree(device_data);
}

void free_from_device(double* device_data) {
    hipFree(device_data);
}

