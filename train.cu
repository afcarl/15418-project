#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>

#include "CycleTimer.h"

#define N 513

__global__ void split() {
}

void data_to_device(double** device_input_data, double* input_data, int size) {
    hipMalloc(device_input_data, sizeof(double) * size);
    hipMemcpy(*device_input_data, input_data, sizeof(double) * size, hipMemcpyHostToDevice);
}

void data_to_device(int** device_input_data, int* input_data, int size) {
    hipMalloc(device_input_data, sizeof(int) * size);
    hipMemcpy(*device_input_data, input_data, sizeof(int) * size, hipMemcpyHostToDevice);
}

void free_from_device(int* device_data) {
    hipFree(device_data);
}

void free_from_device(double* device_data) {
    hipFree(device_data);
}

// From http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
__global__ void init(unsigned int seed, hiprandState_t* states) {
     hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
}

__global__ void randoms(hiprandState_t* states, int* numbers) {
    numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % N;
}

void bootstrap_sample(int** device_nums) {
    hipMalloc(device_nums, sizeof(int) * N);

    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

    init<<<N, 1>>>(time(0), states);
    randoms<<<N, 1>>>(states, *device_nums);

    hipFree(states);
}
